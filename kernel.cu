#include "hip/hip_runtime.h"
﻿#include <hip/hip_runtime.h>
#include ""
#include "hip/device_functions.h"
#include <vector> //for vector  
#include <algorithm> //for generate
#include <cassert>
#include <cstdlib>
#include <iterator>
#include <hipcub/hipcub.hpp>
#include <iostream>
#include <stdio.h>
#include <windows.h> //winapi header  

using namespace std;

// method used while debugging for printing an array
__global__ void printArray(int* a, int n) {
    for (int i = 0; i < n; i++) {
        printf("%d ", a[i]);
    }
    printf("\n");
}

void debugArray(char a[], int* arr, int n) {
    printf("DEBUGGING %s\n", a);
    printArray << <1, 1 >> > (arr, n);
}

// creating a histogram 
__global__ void createHistogram(int* a, int* h, int N)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= N) return;

    int pos = a[tid];
    atomicAdd(&h[pos], 1);

}

// function created to get the prefix-sum of an array using the inclusive scan method
void prefix_sum_on_gpu(int* data, int* output, int size) {
    void* d_temp_storage = NULL;
    size_t   temp_storage_bytes = 0;
    hipcub::DeviceScan::InclusiveSum(d_temp_storage, temp_storage_bytes, data, output, size);
    // Allocate temporary storage for inclusive prefix sum
    hipMalloc(&d_temp_storage, temp_storage_bytes);
    // Run inclusive prefix sum
    hipcub::DeviceScan::InclusiveSum(d_temp_storage, temp_storage_bytes, data, output, size);
    //printf("Successfully prefixed sum");
}

int main()
{
    //N = number of elements
    //M = maximum element + 1 (histogram size)
    constexpr int N = 15, M = 100;

    size_t bytesN = sizeof(int) * N;
    size_t bytesM = sizeof(int) * M;

    vector<int> x(N), s(N);
    generate(x.begin(), x.end(), [M]() {return rand() % M; });

    int* d_x, * d_a, * d_ap, * d_b, * d_bp;

    /*------------------------START OF THE GPU COMPUTATION--------------------------*/

    // x is input arr
    // d_x is copy of x on gpu
    hipMalloc(&d_x, bytesN);
    hipMemcpy(d_x, x.data(), bytesN, hipMemcpyHostToDevice);

    int numThreads = 2;
    int numBlocks = (N + numThreads - 1) / numThreads;
    int numBlocks2 = (M + numThreads - 1) / numThreads;

    hipMalloc(&d_a, bytesM);
    //d_a is histogram of d_x 
    createHistogram << <numThreads, numBlocks >> > (d_x, d_a, N);

    hipMalloc(&d_ap, bytesM);
    //d_ap is prefix sum of d_a
    prefix_sum_on_gpu(d_a, d_ap, M);

    hipMalloc(&d_b, bytesN);
    //d_b is histogram of d_ap
    createHistogram << <numThreads, numBlocks2 >> > (d_ap, d_b, M);

    hipMalloc(&d_bp, bytesN);
    //d_bp is the prefix-sum of db
    prefix_sum_on_gpu(d_b + 1, d_bp + 1, N);

    hipMemcpy(s.data(), d_bp, bytesN, hipMemcpyDeviceToHost);

    /*------------------------END OF THE GPU COMPUTATION--------------------------*/

    // x is the initial array and s is the sorted array.
    for (auto& element : x) cout << element << " ";
    cout << endl;

    for (auto& element : s) cout << element << " ";
    cout << endl;
    return 0;
}